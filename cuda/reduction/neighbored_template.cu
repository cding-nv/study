#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream> 

#if 0
__global__ void reduceCompleteUnrollWarps8 (int *g_idata, int *g_odata,
        unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int b1 = g_idata[idx + 4 * blockDim.x];
        int b2 = g_idata[idx + 5 * blockDim.x];
        int b3 = g_idata[idx + 6 * blockDim.x];
        int b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}
#endif

template <unsigned int iBlockSize>
__global__ void reduceCompleteUnroll(int *g_idata, int *g_odata,
                                     unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int b1 = g_idata[idx + 4 * blockDim.x];
        int b2 = g_idata[idx + 5 * blockDim.x];
        int b3 = g_idata[idx + 6 * blockDim.x];
        int b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (iBlockSize >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (iBlockSize >= 512 && tid < 256)  idata[tid] += idata[tid + 256];

    __syncthreads();

    if (iBlockSize >= 256 && tid < 128)  idata[tid] += idata[tid + 128];

    __syncthreads();

    if (iBlockSize >= 128 && tid < 64)   idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


#define BLOCK 2

int main() {
    int h[] = {13, 27, 15, 14, 33, 2, 24, 6};
    const auto count = sizeof(h) / sizeof(h[0]);
    const int size = count * sizeof(int);

    int* d;

    hipMalloc(&d, size);
    hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    int* o_d;
    hipMalloc(&o_d, BLOCK * sizeof(int));

    reduceUnrolling2 <<<BLOCK, count / BLOCK >>>(d, o_d, size);

    int result[BLOCK];
    hipMemcpy(&result, o_d, BLOCK * sizeof(int), hipMemcpyDeviceToHost);

    int sum = 0;
    for (int i = 0; i < BLOCK; i++) {
        sum += result[i];
        std::cout << "block " << i << " = " << result[i] << std::endl;

    }
    std::cout << "Sum = " << result[0] + result[1] << std::endl;


    //getchar();

    hipFree(d);
    //delete[] h;

    return 0;
}

