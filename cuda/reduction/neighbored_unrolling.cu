#include "hip/hip_runtime.h"
#include <iostream> 

#if 0
__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if(idx >= n) return;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}
#endif

__global__ void reduceUnrolling2 (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // unrolling 2
    if (idx + blockDim.x < n) g_idata[idx] += g_idata[idx + blockDim.x];

    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


#define BLOCK 2

int main() {
    int h[] = {13, 27, 15, 14, 33, 2, 24, 6};
    const auto count = sizeof(h) / sizeof(h[0]);
    const int size = count * sizeof(int);

    int* d;

    hipMalloc(&d, size);
    hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    int* o_d;
    hipMalloc(&o_d, BLOCK * sizeof(int));

    reduceUnrolling2 <<<BLOCK, count / BLOCK >>>(d, o_d, size);

    int result[BLOCK];
    hipMemcpy(&result, o_d, BLOCK * sizeof(int), hipMemcpyDeviceToHost);

    int sum = 0;
    for (int i = 0; i < BLOCK; i++) {
        sum += result[i];
        std::cout << "block " << i << " = " << result[i] << std::endl;

    }
    std::cout << "Sum = " << result[0] + result[1] << std::endl;


    //getchar();

    hipFree(d);
    //delete[] h;

    return 0;
}

