#include "hip/hip_runtime.h"
#include <iostream> 

__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if(idx >= n) return;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }

        __syncthreads();   // We don't need this when "s <= 32", instruction are SIMD synchronous within a warp
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

#define BLOCK 2

int main() {
    int h[] = {13, 27, 15, 14, 33, 2, 24, 6};
    const auto count = sizeof(h) / sizeof(h[0]);
    const int size = count * sizeof(int);

    int* d;

    hipMalloc(&d, size);
    hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    int* o_d;
    hipMalloc(&o_d, BLOCK * sizeof(int));

    reduceInterleaved <<<BLOCK, count / BLOCK >>>(d, o_d, size);

    int result[BLOCK];
    hipMemcpy(&result, o_d, BLOCK * sizeof(int), hipMemcpyDeviceToHost);

    int sum = 0;
    for (int i = 0; i < BLOCK; i++) {
        sum += result[i];
        std::cout << "block " << i << " = " << result[i] << std::endl;

    }
    std::cout << "Sum = " << result[0] + result[1] << std::endl;


    //getchar();

    hipFree(d);
    //delete[] h;

    return 0;
}

