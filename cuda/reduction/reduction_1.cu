#include "hip/hip_runtime.h"


#include <iostream>
#include <numeric>
using namespace std;

__global__ void sum(int* input)
{
	const int tid = threadIdx.x;

	auto step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			const auto fst = tid * step_size * 2;
			const auto snd = fst + step_size;
			input[fst] += input[snd];
            printf("tid=%d, fst=%d, snd=%d, step_size=%d\n", tid, fst, snd, step_size);
		}

		step_size <<= 1; 
		number_of_threads >>= 1;
	}
}

int main()
{
	int h[] = {13, 27, 15, 14, 33, 2, 24, 6};
    const auto count = sizeof(h) / sizeof(h[0]);
    const int size = count * sizeof(int);

	int* d;
	
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	sum <<<1, count / 2 >>>(d);

	int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "Sum is " << result << endl;

	getchar();

	hipFree(d);
	//delete[] h;

	return 0;
}
