#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 5

#define intswap(A,B) {int temp = A; A = B; B = temp;}

__global__ void sort(int *c,int *count) {
    int l;
    if(*count % 2 == 0)
        l = *count / 2;
    else
        l=(*count / 2) + 1;
    for(int i = 0; i < l; i++) {
        if((!(threadIdx.x & 1)) && (threadIdx.x < (*count - 1))) { //even phase
            if(c[threadIdx.x] > c[threadIdx.x + 1])
                intswap(c[threadIdx.x], c[threadIdx.x + 1]);
            printf("even threadIdx %d \n", threadIdx.x);
	}

        __syncthreads();
        if((threadIdx.x & 1) && (threadIdx.x < (*count - 1))) {   //odd phase
            if(c[threadIdx.x] > c[threadIdx.x + 1])
                intswap(c[threadIdx.x], c[threadIdx.x + 1]);
            printf("odd threadIdx %d \n", threadIdx.x);
        }
        __syncthreads();
    }//for
}

// https://stackoverflow.com/questions/29581115/odd-even-sort-using-cuda-programming

int main() {
    int a[N] = {8, 6, 2, 3, 7};
    int b[N];
    int n = N;

#if 0
    printf("enter size of array");
    scanf("%d",&n);
    if (n > N) {
        printf("too large!\n");
        return 1;
    }
    printf("enter the elements of array");
    for(int i=0; i<n; i++) {
        scanf("%d",&a[i]);
    }
#endif
    printf("ORIGINAL ARRAY : \n");
    for(int i = 0; i < n; i++) {
        printf("%d ",a[i]);
    }
    printf("\n");

    int *c, *count;
    hipMalloc((void**)&c, sizeof(int) * N);
    hipMalloc((void**)&count, sizeof(int));
    hipMemcpy(c, &a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(count, &n, sizeof(int), hipMemcpyHostToDevice);
    sort<<< 1, n >>>(c, count);
    hipMemcpy(&b, c, sizeof(int) * N, hipMemcpyDeviceToHost);
    printf("\nSORTED ARRAY : \n");
    for(int i = 0; i < n; i++) {
        printf("%d ",b[i]);
    }

    printf("\n");
}
