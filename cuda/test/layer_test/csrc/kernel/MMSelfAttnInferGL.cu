#include "hip/hip_runtime.h"
#include "MMSelfAttnInferGL.h"
#include "BatchGemm.h"
#include "Context.h"
#include "MMSelfAttnCom.h"
#include "Softmax.h"
#include "UniOps.h"


template <typename T>
void
MMSelfAttnInferGL::forward(
        hipStream_t stream0,
        const int64_t* modal_index,
        const T* query_layer,
        const T* key_layer,
        const T* value_layer,
        const T* local_attention_mask,
        const T* global_k,
        const T* global_v,
        const int64_t* global_selection_padding_mask_zeros,
        T* context_layer,
        T* buffer,
        int modal_cnt,
        int batch_size,
        int seq_len,
        int num_attention_heads,
        int attention_head_size,
        int max_num_global_indices_per_batch,
        int global_selection_padding_mask_zeros_nRow,
        bool use_multistream,
        bool fast_fp32) {
    MMSelfAttnCom com(modal_index, modal_cnt, batch_size, num_attention_heads, attention_head_size, max_num_global_indices_per_batch);
    const float alpha = 1. / sqrt(attention_head_size);

    for (int i = 0; i < modal_cnt; ++i) {
        hipStream_t stream = use_multistream ? Context::instance().getStream(i) : stream0;
        hipblasHandle_t handle = Context::instance().getHandle(stream, fast_fp32);

        const int begin = com.get_index_begin(i);
        const int cur_seq_len = com.get_current_seq_len(i);
        const int cur_l_qkv_numel = com.get_current_local_qkv_numel(i);
        const int cur_l_attn_numel_align = com.get_current_local_attn_numel_align<T>(i);
        const int cur_g_attn_numel_align = com.get_current_global_attn_numel_align<T>(i);
        const int cur_attn_numel_align = com.get_current_attn_numel_align<T>(i);

        T* buff_A = buffer;                                 // cur_l_attn_numel_align
        T* buff_B = buffer + cur_l_attn_numel_align;        // max(cur_l_qkv_numel, cur_g_attn_numel_align)
        T* buff_C = buffer + cur_l_attn_numel_align + max(cur_l_qkv_numel, cur_g_attn_numel_align);     // // max(cur_l_qkv_numel, cur_attn_numel_align)
        if (use_multistream) {
            buffer += cur_l_attn_numel_align + max(cur_l_qkv_numel, cur_g_attn_numel_align) + max(cur_l_qkv_numel, cur_attn_numel_align);
        }

        SlicePermute<T>::run(
                begin,
                seq_len,
                query_layer,
                batch_size,
                num_attention_heads,
                cur_seq_len,
                attention_head_size,
                buff_C,                             // C0, cur_l_qkv_numel, out
                stream);

        SlicePermute<T>::run(
                begin,
                seq_len,
                key_layer,
                batch_size,
                num_attention_heads,
                cur_seq_len,
                attention_head_size,
                buff_B,                             // B0, cur_l_qkv_numel, out
                stream);

        SliceExpand12<T>::run(
                begin,
                seq_len,
                local_attention_mask,
                batch_size,
                num_attention_heads,
                cur_seq_len,
                cur_seq_len,
                buff_A,                             // A0, cur_l_attn_numel_align, out
                stream);

        BatchGemm<T, false, true>::forwardA(
                buff_C,                             // C0, cur_l_qkv_numel, in
                buff_B,                             // B0, cur_l_qkv_numel, in
                buff_A,                             // A0, cur_l_attn_numel_align, inout
                handle,
                batch_size * num_attention_heads,
                cur_seq_len,
                cur_seq_len,
                attention_head_size,
                alpha);

        BatchGemm<T, false, true>::forward(
                buff_C,                             // C0, cur_l_qkv_numel, in
                global_k,
                buff_B,                             // B1, cur_g_attn_numel_align, out
                handle,
                batch_size * num_attention_heads,
                cur_seq_len,
                max_num_global_indices_per_batch,
                attention_head_size,
                alpha);

        if (global_selection_padding_mask_zeros_nRow) {
            CopySlice<T>::run(
                    num_attention_heads,
                    cur_seq_len,
                    max_num_global_indices_per_batch,
                    buff_B,                         // B1, cur_g_attn_numel_align, inout
                    global_selection_padding_mask_zeros_nRow,
                    global_selection_padding_mask_zeros,
                    -10000.f,
                    stream);
        }

        Cat<T>::run(
                batch_size * num_attention_heads * cur_seq_len,
                max_num_global_indices_per_batch,
                buff_B,                             // B1, cur_g_attn_numel_align, in
                cur_seq_len,
                buff_A,                             // A0, cur_l_attn_numel_align, in
                buff_C,                             // C1, cur_attn_numel_align, out
                stream);

        Softmax<T>::forward(
                buff_C,                             // C1, cur_attn_numel_align, inout
                batch_size * num_attention_heads * cur_seq_len,
                max_num_global_indices_per_batch + cur_seq_len,
                stream);

        Narrow<T>::run(
                batch_size * num_attention_heads * cur_seq_len,
                buff_C,                             // C1, cur_attn_numel_align, in
                max_num_global_indices_per_batch,
                buff_B,                             // B2, cur_g_attn_numel_align, out
                cur_seq_len,
                buff_A,                             // A1, cur_l_attn_numel_align, out
                stream);

        BatchGemm<T, false, false>::forward(
                buff_B,                             // B2, cur_g_attn_numel_align, in
                global_v,
                buff_C,                             // C2, cur_l_qkv_numel, out
                handle,
                batch_size * num_attention_heads,
                cur_seq_len,
                attention_head_size,
                max_num_global_indices_per_batch);

        SlicePermute<T>::run(
                begin,
                seq_len,
                value_layer,
                batch_size,
                num_attention_heads,
                cur_seq_len,
                attention_head_size,
                buff_B,                             // B3, cur_l_qkv_numel, out
                stream);

        BatchGemm<T, false, false>::forwardA(
                buff_A,                             // A1, cur_l_attn_numel_align, in
                buff_B,                             // B3, cur_l_qkv_numel, in
                buff_C,                             // C2, cur_l_qkv_numel, inout
                handle,
                batch_size * num_attention_heads,
                cur_seq_len,
                attention_head_size,
                cur_seq_len);

        PermuteSlice<T>::run(
                begin,
                seq_len,
                context_layer,                      // local_qkv_numel, out
                batch_size,
                num_attention_heads,
                cur_seq_len,
                attention_head_size,
                buff_C,                             // C2, cur_l_qkv_numel, in
                stream);
    }
}


template
void
MMSelfAttnInferGL::forward(
        hipStream_t stream0,
        const int64_t* modal_index,
        const float* query_layer,
        const float* key_layer,
        const float* value_layer,
        const float* local_attention_mask,
        const float* global_k,
        const float* global_v,
        const int64_t* global_selection_padding_mask_zeros,
        float* context_layer,
        float* buffer,
        int modal_cnt,
        int batch_size,
        int seq_len,
        int num_attention_heads,
        int attention_head_size,
        int max_num_global_indices_per_batch,
        int global_selection_padding_mask_zeros_nRow,
        bool use_multistream,
        bool fast_fp32);


template
void
MMSelfAttnInferGL::forward(
        hipStream_t stream0,
        const int64_t* modal_index,
        const __half* query_layer,
        const __half* key_layer,
        const __half* value_layer,
        const __half* local_attention_mask,
        const __half* global_k,
        const __half* global_v,
        const int64_t* global_selection_padding_mask_zeros,
        __half* context_layer,
        __half* buffer,
        int modal_cnt,
        int batch_size,
        int seq_len,
        int num_attention_heads,
        int attention_head_size,
        int max_num_global_indices_per_batch,
        int global_selection_padding_mask_zeros_nRow,
        bool use_multistream,
        bool fast_fp32);

