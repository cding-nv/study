#include "hip/hip_runtime.h"
#include "general_kernels.h"
#include "common_op.h"

#define WARP_SIZE 32

namespace cg = cooperative_groups;

/*
Fused bias add, residual (elementwise) add, and normalization layer.

For FP16, this kernel does not promote to FP32 in order to utilize the 2x throughput for
half2 instructions, and avoid the conversion overhead (1/8 of __hal2 arithmetic).

For specific launch constraints, see the launch functions.
*/

#define NORM_REG (MAX_REGISTERS / 4)  // 64

__global__ void fused_bias_residual_layer_norm(float* vals,
                                               const float* residual,
                                               const float* gamma,
                                               const float* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               float* vars,
                                               float* means,
                                               int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id / WARP_SIZE;

    float vals_arr[NORM_REG];
    __shared__ float shr[MAX_WARP_NUM];

    residual += (row * row_stride);
    vals += (row * row_stride);

    float sum = 0.f;
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = residual[i * iteration_stride + id];
        sum += vals_arr[i];
    }
    if (high_index < row_stride) {
        vals_arr[iterations] = residual[high_index];
        sum += vals_arr[iterations];
        iterations++;
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) sum = shr[g.thread_rank()];

#if !defined(__STOCHASTIC_MODE__) || __CUDA_ARCH__ < 700
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { sum += g.shfl_down(sum, i); }

    sum = g.shfl(sum, 0);
    float mean = sum / row_stride;
    if (training)
        if (g.thread_rank() == 0) means[row] = mean;
    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] -= mean;
        variance += vals_arr[i] * vals_arr[i];
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { variance += g.shfl_down(variance, i); }
    variance = g.shfl(variance, 0);
    variance /= row_stride;
    variance += epsilon;
    if (training)
        if (g.thread_rank() == 0) vars[row] = variance;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = vals_arr[i] * rsqrtf(variance);
        vals_arr[i] =
            vals_arr[i] * gamma[i * iteration_stride + id] + beta[i * iteration_stride + id];
        vals[i * iteration_stride + id] = vals_arr[i];
    }
    if ((high_index) < row_stride) {
        vals_arr[iterations] = vals_arr[iterations] * rsqrtf(variance);
        vals_arr[iterations] = vals_arr[iterations] * gamma[high_index] + beta[high_index];
        vals[high_index] = vals_arr[iterations];
    }
}

__global__ void fused_bias_residual_layer_norm(half* vals,
                                               const half* residual,
                                               const half* gamma,
                                               const half* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               half* vars,
                                               half* means,
                                               int row_stride)
{
#if __CUDA_ARCH__ >= 700
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id >> 5;

    float2 vals_f[NORM_REG];
    __shared__ float shr[MAX_WARP_NUM];

    half2* vals_cast = reinterpret_cast<half2*>(vals);
    const half2* residual_cast = reinterpret_cast<const half2*>(residual);

    residual_cast += (row * row_stride);
    vals_cast += (row * row_stride);

    float sum = 0.f;
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_f[i] = __half22float2(residual_cast[i * iteration_stride + id]);
        sum += vals_f[i].x;
        sum += vals_f[i].y;
    }
    if ((high_index) < row_stride) {
        vals_f[iterations] = __half22float2(residual_cast[high_index]);
        sum += vals_f[iterations].x;
        sum += vals_f[iterations].y;
        iterations++;
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) sum = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { sum += g.shfl_down(sum, i); }
    sum = g.shfl(sum, 0);
    float mean = sum / (row_stride * 2);

    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        vals_f[i].x -= mean;
        vals_f[i].y -= mean;
        variance += vals_f[i].x * vals_f[i].x;
        variance += vals_f[i].y * vals_f[i].y;
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { variance += g.shfl_down(variance, i); }
    variance = g.shfl(variance, 0);
    variance /= (row_stride * 2);
    variance += epsilon;

    half2 variance_h = __float2half2_rn(variance);
    const half2* gamma_cast = reinterpret_cast<const half2*>(gamma);
    const half2* beta_cast = reinterpret_cast<const half2*>(beta);

    if (training && g.thread_rank() == 0) {
        vars[row] = __float2half(variance);
        means[row] = __float2half(mean);
    }
    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        half2 vals_arr = __float22half2_rn(vals_f[i]);
        vals_arr = vals_arr * h2rsqrt(variance_h);
        vals_arr =
            vals_arr * gamma_cast[i * iteration_stride + id] + beta_cast[i * iteration_stride + id];
        vals_cast[i * iteration_stride + id] = vals_arr;
    }
    if ((high_index) < row_stride) {
        half2 vals_arr = __float22half2_rn(vals_f[iterations]);
        vals_arr = vals_arr * h2rsqrt(variance_h);
        vals_arr = vals_arr * gamma_cast[high_index] + beta_cast[high_index];
        vals_cast[high_index] = vals_arr;
    }
#endif
}

template <typename T>
void launch_bias_residual_layer_norm(T* vals,
                                     const T* residual,
                                     const T* gamma,
                                     const T* beta,
                                     float epsilon,
                                     int batch_size,
                                     int hidden_dim,
                                     hipStream_t stream,
                                     bool preLayerNorm,
                                     bool training,
                                     T* vars,
                                     T* means);

template <>
void launch_bias_residual_layer_norm<float>(float* vals,
                                            const float* residual,
                                            const float* gamma,
                                            const float* beta,
                                            float epsilon,
                                            int batch_size,
                                            int hidden_dim,
                                            hipStream_t stream,
                                            bool preLayerNorm,
                                            bool training,
                                            float* vars,
                                            float* means)
{
    int threads = THREADS;

    dim3 grid_dim(batch_size);

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim(threads);

    fused_bias_residual_layer_norm<<<grid_dim, block_dim, 0, stream>>>(
        vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means, hidden_dim);
}

template <>
void launch_bias_residual_layer_norm<half>(half* vals,
                                             const half* residual,
                                             const half* gamma,
                                             const half* beta,
                                             float epsilon,
                                             int batch_size,
                                             int hidden_dim,
                                             hipStream_t stream,
                                             bool preLayerNorm,
                                             bool training,
                                             half* vars,
                                             half* means)
{
    int threads = 128;

    dim3 grid_dim(batch_size);

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim(threads);

    fused_bias_residual_layer_norm<<<grid_dim, block_dim, 0, stream>>>(
        vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means, hidden_dim / 2);
}

__global__ void fused_bias_residual_layer_norm(float* vals,
                                               const float* residual,
                                               const float* gamma,
                                               const float* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               float* vars,
                                               int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id / 32;

    float vals_arr[NORM_REG];
    __shared__ float shr[MAX_WARP_NUM];

    residual += (row * row_stride);
    vals += (row * row_stride);

    float sum = 0.f;
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = residual[i * iteration_stride + id];
        sum += vals_arr[i];
    }
    if ((high_index) < row_stride) {
        vals_arr[iterations] = residual[high_index];
        sum += vals_arr[iterations];
        iterations++;
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) sum = shr[g.thread_rank()];

#if !defined(__STOCHASTIC_MODE__) || __CUDA_ARCH__ < 700
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { sum += g.shfl_down(sum, i); }

    sum = g.shfl(sum, 0);
    float mean = sum / row_stride;
    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] -= mean;
        variance += vals_arr[i] * vals_arr[i];
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { variance += g.shfl_down(variance, i); }
    variance = g.shfl(variance, 0);
    variance /= row_stride;
    variance += epsilon;
    if (training)
        if (g.thread_rank() == 0) vars[row] = variance;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = vals_arr[i] * rsqrtf(variance);
        vals_arr[i] =
            vals_arr[i] * gamma[i * iteration_stride + id] + beta[i * iteration_stride + id];
        vals[i * iteration_stride + id] = vals_arr[i];
    }
    if ((high_index) < row_stride) {
        vals_arr[iterations] = vals_arr[iterations] * rsqrtf(variance);
        vals_arr[iterations] = vals_arr[iterations] * gamma[high_index] + beta[high_index];
        vals[high_index] = vals_arr[iterations];
    }
}

__global__ void fused_bias_residual_layer_norm(half* vals,
                                               const half* residual,
                                               const half* gamma,
                                               const half* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               half* vars,
                                               int row_stride)
{
#if __CUDA_ARCH__ >= 700

    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id >> 5;

    float2 vals_f[NORM_REG];
    __shared__ float shr[MAX_WARP_NUM];

    half2* vals_cast = reinterpret_cast<half2*>(vals);
    const half2* residual_cast = reinterpret_cast<const half2*>(residual);

    residual_cast += (row * row_stride);
    vals_cast += (row * row_stride);

    float sum = 0.f;
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_f[i] = __half22float2(residual_cast[i * iteration_stride + id]);
        sum += vals_f[i].x;
        sum += vals_f[i].y;
    }
    if ((high_index) < row_stride) {
        vals_f[iterations] = __half22float2(residual_cast[high_index]);
        sum += vals_f[iterations].x;
        sum += vals_f[iterations].y;
        iterations++;
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) sum = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { sum += g.shfl_down(sum, i); }
    sum = g.shfl(sum, 0);
    float mean = sum / (row_stride * 2);

    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        vals_f[i].x -= mean;
        vals_f[i].y -= mean;
        variance += vals_f[i].x * vals_f[i].x;
        variance += vals_f[i].y * vals_f[i].y;
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { variance += g.shfl_down(variance, i); }
    variance = g.shfl(variance, 0);
    variance /= (row_stride * 2);
    variance += epsilon;

    half2 variance_h = __float2half2_rn(variance);
    const half2* gamma_cast = reinterpret_cast<const half2*>(gamma);
    const half2* beta_cast = reinterpret_cast<const half2*>(beta);

    if (training && g.thread_rank() == 0) vars[row] = __float2half(variance);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        half2 vals_arr = __float22half2_rn(vals_f[i]);
        vals_arr = vals_arr * h2rsqrt(variance_h);
        vals_arr =
            vals_arr * gamma_cast[i * iteration_stride + id] + beta_cast[i * iteration_stride + id];
        vals_cast[i * iteration_stride + id] = vals_arr;
    }
    if ((high_index) < row_stride) {
        half2 vals_arr = __float22half2_rn(vals_f[iterations]);
        vals_arr = vals_arr * h2rsqrt(variance_h);
        vals_arr = vals_arr * gamma_cast[high_index] + beta_cast[high_index];
        vals_cast[high_index] = vals_arr;
    }
#endif
}

template <typename T>
void launch_bias_residual_layer_norm(T* vals,
                                     const T* residual,
                                     const T* gamma,
                                     const T* beta,
                                     float epsilon,
                                     int batch_size,
                                     int hidden_dim,
                                     hipStream_t stream,
                                     bool preLayerNorm,
                                     bool training,
                                     T* vars);

/*
To tune this launch the following restrictions must be met:

For float:
row_stride == hidden_size
threads * iterations == row_stride
threads is in [32, 64, 128, 256, 512, 1024]

For half:
row_stride == hidden_size / 2
threads * iterations == row_stride
threads is in [32, 64, 128, 256, 512, 1024]

*/

template <>
void launch_bias_residual_layer_norm<float>(float* vals,
                                            const float* residual,
                                            const float* gamma,
                                            const float* beta,
                                            float epsilon,
                                            int batch_size,
                                            int hidden_dim,
                                            hipStream_t stream,
                                            bool preLayerNorm,
                                            bool training,
                                            float* vars)
{
    int threads = THREADS;

    dim3 grid_dim(batch_size);

    // There are some limitations to call below functions, now just enumerate the situations.

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim(threads);

    fused_bias_residual_layer_norm<<<grid_dim, block_dim, 0, stream>>>(
        vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, hidden_dim);
}

template <>
void launch_bias_residual_layer_norm<half>(half* vals,
                                             const half* residual,
                                             const half* gamma,
                                             const half* beta,
                                             float epsilon,
                                             int batch_size,
                                             int hidden_dim,
                                             hipStream_t stream,
                                             bool preLayerNorm,
                                             bool training,
                                             half* vars)
{
    int threads = 128;

    dim3 grid_dim(batch_size);

    // There are some limitations to call below functions, now just enumerate the situations.

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim(threads);
    fused_bias_residual_layer_norm<<<grid_dim, block_dim, 0, stream>>>(
        vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, hidden_dim / 2);
}

/* Normalize Gamma & Betta gradients
 * Compute gradients using either X_hat or
 * normalize input (invertible).
 * Combine transpose with gradients computation.
 */

template <typename T>
__global__ void LayerNormBackward1(const T* __restrict__ out_grad,
                                   const T* __restrict__ vals_hat,
                                   const T* __restrict__ gamma,
                                   const T* __restrict__ betta,
                                   T* __restrict__ gamma_grad,
                                   T* __restrict__ betta_grad,
                                   int rows,
                                   int width,
                                   bool invertible)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    float betta_reg = (invertible ? (float)betta[idx] : 0.0f);
    float gamma_reg = (float)gamma[idx];

    // Loop across matrix height
    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad[offset];
        float val = (invertible ? ((float)vals_hat[offset] - betta_reg) / gamma_reg
                                : (float)vals_hat[offset]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        int pos = blockIdx.x * TILE_DIM + threadIdx.y;
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}

/* Normalize Gamma & Betta gradients
 * Compute gradients using the input to
 * the normalize.
 * Combine transpose with gradients computation.
 */

// dishengbin, back1
// grid (hidden_dim/TILE_DIM, 1024)
// block (TILE_DIM)
// output: (32*TILE_DIM, hidden_dim)
template <typename T>
__global__ void LayerNormBackward1_0(const T* __restrict__ out_grad,
                                   const T* __restrict__ X_data,
                                   const T* __restrict__ vars,
                                   const T* __restrict__ means,
                                   T* __restrict__ gamma_inter,
                                   T* __restrict__ betta_inter,
                                   int rows,
                                   int width)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x; // hidden_dim

    float betta_tmp = 0.0f;
    float gamma_tmp = 0.0f;

    for (int idy = blockIdx.y; idy < rows; idy += gridDim.y) {
        int id = idy * width + idx;

        float grad = 0.0f;
        float val = 0.0f;

        if (idx < width) {
            grad = (float)out_grad[id];
            val = (float)X_data[id];
        }
        val = (val - (float)means[idy]) * rsqrtf((float)vars[idy]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);
    }

    if (idx < width) {
        int id = blockIdx.y * width + idx;
        betta_inter[id] = betta_tmp;
        gamma_inter[id] = gamma_tmp;
    }
}

// grid_dim3 (hidden_dim/TILE)
// block_dim3 (TILE, TILE)
template <typename T>
__global__ void LayerNormBackward1_1(
        T* __restrict__ gamma_inter,
        T* __restrict__ betta_inter,
        T* __restrict__ gamma_grad,
        T* __restrict__ betta_grad,
        int rows,
        int width
        ) {
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    int pos = blockIdx.x * TILE_DIM + threadIdx.y;

    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float gamma = (float)gamma_inter[offset];
        float betta = (float)betta_inter[offset];

        betta_tmp += betta;
        gamma_tmp += gamma;

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

    __syncthreads();

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0 && pos < width) {
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}


// dishengbin, back1
template <typename T>
__global__ void LayerNormBackward1(const T* __restrict__ out_grad,
                                   const T* __restrict__ X_data,
                                   const T* __restrict__ vars,
                                   const T* __restrict__ means,
                                   T* __restrict__ gamma_grad,
                                   T* __restrict__ betta_grad,
                                   int rows,
                                   int width)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    int pos = blockIdx.x * TILE_DIM + threadIdx.y;
    // Loop across matrix height

    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = 0.0f;
        float val = 0.0f;
        if (idx < width) {
            grad = (float)out_grad[offset];
            val = (float)X_data[offset];
        }
        val = (val - (float)means[r]) * rsqrtf((float)vars[r]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0 && pos < width) {
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}


/* Backward Normalize (Input-Gradient)
 * Using the means and variances from the input
 * This type of backward is invertible!
 * We do the backward using the X_hat (X - u) / sqrt(variance) or the output of Normalization.
 */

__global__ void LayerNormBackward2(const float* out_grad,
                                   const float* vals_hat,
                                   const float* gamma,
                                   const float* betta,
                                   const float* vars,
                                   float* inp_grad,
                                   bool invertible,
                                   int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = (THREADS < row_stride ? THREADS : row_stride) / WARP_SIZE;
    __shared__ float partialSum[MAX_WARP_NUM];

    out_grad += (row * row_stride);
    vals_hat += (row * row_stride);
    inp_grad += (row * row_stride);

    float vals_arr[NORM_REG];
    float vals_hat_arr[NORM_REG];
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] =
            (invertible ? (vals_hat[i * iteration_stride + id] - betta[i * iteration_stride + id]) /
                              gamma_reg
                        : vals_hat[i * iteration_stride + id]);
    }
    if ((high_index) < row_stride) {
        float gamma_reg = gamma[high_index];
        vals_arr[iterations] = out_grad[high_index];
        vals_arr[iterations] *= gamma_reg;
        vals_hat_arr[iterations] =
            (invertible ? (vals_hat[high_index] - betta[high_index]) / gamma_reg
                        : vals_hat[high_index]);
        iterations++;
    }

    float var_reg = vars[row];

    float sum = 0;
    for (int i = 0; i < iterations; i++) {
        sum += vals_hat_arr[i] * vals_arr[i] *
               sqrtf(var_reg);           // dval_hat = gamma * (x - u) * out_grad
        vals_arr[i] *= rsqrtf(var_reg);  // dvar_inv = gamma * out_grad / sqrt(var)
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) { vals_arr[i] += ((-sum * vals_hat_arr[i]) / var_reg); }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) inp_grad[i * iteration_stride + id] = (vals_arr[i] - sum);
    if ((high_index) < row_stride) inp_grad[high_index] = (vals_arr[iterations] - sum);
}

__global__ void LayerNormBackward2(const half* out_grad,
                                   const half* vals_hat,
                                   const half* gamma,
                                   const half* betta,
                                   const half* vars,
                                   half* inp_grad,
                                   bool invertible,
                                   int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = (iteration_stride < row_stride ? iteration_stride : row_stride) / WARP_SIZE;
    __shared__ float partialSum[MAX_WARP_NUM];

    half2 vals_arr[NORM_REG];
    float2 vals_arr_f[NORM_REG];
    half2 vals_hat_arr[NORM_REG];

    half2* inp_grad_h = reinterpret_cast<half2*>(inp_grad);
    const half2* out_grad_h = reinterpret_cast<const half2*>(out_grad);
    const half2* vals_hat_h = reinterpret_cast<const half2*>(vals_hat);

    inp_grad_h += (row * row_stride);
    out_grad_h += (row * row_stride);
    vals_hat_h += (row * row_stride);

    const half2* gamma_h = reinterpret_cast<const half2*>(gamma);
    const half2* betta_h = (invertible ? reinterpret_cast<const half2*>(betta) : nullptr);
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] =
            (invertible
                 ? (vals_hat_h[i * iteration_stride + id] - betta_h[i * iteration_stride + id]) /
                       gamma_reg
                 : vals_hat_h[i * iteration_stride + id]);
    }
    if ((high_index) < row_stride) {
        half2 gamma_reg = gamma_h[high_index];
        vals_arr[iterations] = out_grad_h[high_index];
        vals_arr[iterations] *= gamma_reg;
        vals_hat_arr[iterations] =
            (invertible ? (vals_hat_h[high_index] - betta_h[high_index]) / gamma_reg
                        : vals_hat_h[high_index]);
        iterations++;
    }
    half var_h = vars[row];
    half2 var_reg = __halves2half2(var_h, var_h);

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        half2 result_h = (vals_hat_arr[i] * vals_arr[i] * h2sqrt(var_reg));
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        half2 temp = ((-sum_h * vals_hat_arr[i]) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 temp_f = __half22float2(temp);
        vals_arr_f[i].x += temp_f.x;
        vals_arr_f[i].y += temp_f.y;
    }
    sum = 0.f;

    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        half2 temp = __float22half2_rn(vals_arr_f[i]);

        inp_grad_h[i * iteration_stride + id] = temp;
    }
    if ((high_index) < row_stride) {
        vals_arr_f[iterations].x -= sum;
        vals_arr_f[iterations].y -= sum;
        half2 temp = __float22half2_rn(vals_arr_f[iterations]);

        inp_grad_h[high_index] = temp;
    }
}

template <>
void launch_layerNorm_backward<float>(const float* out_grad,
                                      const float* vals_hat,
                                      const float* vars,
                                      const float* gamma,
                                      float* gamma_grad,
                                      float* betta_grad,
                                      float* inp_grad,
                                      int batch,
                                      int hidden_dim,
                                      hipStream_t stream,
                                      bool invertible,
                                      const float* betta)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream>>>(
        out_grad, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads);

    LayerNormBackward2<<<grid_dim2, block_dim2, 0, stream>>>(
        out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible, hidden_dim);
}

template <>
void launch_layerNorm_backward<half>(const half* out_grad,
                                       const half* vals_hat,
                                       const half* vars,
                                       const half* gamma,
                                       half* gamma_grad,
                                       half* betta_grad,
                                       half* inp_grad,
                                       int batch,
                                       int hidden_dim,
                                       hipStream_t stream,
                                       bool invertible,
                                       const half* betta)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<half><<<grid_dim, block_dim, 0, stream>>>(
        out_grad, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads / 2);

    LayerNormBackward2<<<grid_dim2, block_dim2, 0, stream>>>(
        out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible, hidden_dim / 2);
}

/* Backward Normalize (Input-Gradient)
 * Using the means and variances from the input
 * This type of backward is not invertible!
 * We do the backward using the input (X)
 */
#define FINAL_MASK 0xffffffff
template <typename T>
__inline__ __device__
T warpReduceSum(T val)
{
  #pragma unroll
  for(int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
  return val;
}

__global__ void LayerNormBackward2_32(const float* out_grad,
                                   const float* X_vals,
                                   const float* gamma,
                                   const float* vars,
                                   const float* means,
                                   float* inp_grad,
                                   int row_stride,
                                   int batch,
                                   int head_num,
                                   int seq_len,
                                   float alpha)
{
    //__shared__ float first[8];
    //__shared__ float second[8];
    //int tid = threadIdx.x;
    int wid = threadIdx.x  / WARP_SIZE;
    // id for [h*N, T, C/h]
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    // id for [C/h]
    int laneId = threadIdx.x % WARP_SIZE;
    // id for [h*N, T]
    //int gwid = wid + blockIdx.x * blockDim.x / WARP_SIZE;
    int gwid = gid / WARP_SIZE;

    float outgrad = out_grad[gid];
    float gm = gamma[laneId];

    float dlxhat = outgrad * gm;
    float x_val = X_vals[gid];
    float xmu = x_val - means[gwid];

    float xivar = rsqrtf(vars[gwid] + 1.0e-12); // 1/tf.sqrt(x_var + epsilon)
    float tmp = dlxhat * xmu * xivar * xivar * xivar;

    // [h*N, T]
    float dlvar = warpReduceSum(tmp);
    //if (laneId == 0)
        //first[wid] = dlvar;
    //__syncthreads();

    float dvarx = xmu / row_stride;
    float dlmu =  0.0f - dlxhat * xivar; // [h*N, T, C/h] 
    float dlmu_sum = warpReduceSum(dlmu); // [h*N, T]
    //if (laneId == 0)
        //second[wid] = dlmu_sum;
    //__syncthreads();

    float dout = dlxhat * xivar - dlvar * dvarx + dlmu_sum / row_stride;

    int seq_id = gwid % seq_len;
    int batch_id = gwid / seq_len % batch;
    int head_id = gwid / (seq_len * batch);
    int out_id = laneId + row_stride * (head_id + seq_id * head_num + batch_id * head_num * seq_len);

    if (x_val < 0.0f)
        dout = alpha * dout;

    inp_grad[out_id] = dout;
}

// The result is incorrect
//__global__ void LayerNormBackward2_32(const float* out_grad,
//                                   const float* X_vals,
//                                   const float* gamma,
//                                   const float* vars,
//                                   const float* means,
//                                   float* inp_grad,
//                                   int row_stride,
//                                   int batch,
//                                   int head_num,
//                                   int seq_len,
//                                   float alpha)
//{
//    //__shared__ float first[32];
//    //__shared__ float second[32];
//    //int wid = threadIdx.x  / WARP_SIZE;
//    // id for [h*N, T, C/h]
//    //int gid = threadIdx.x + blockIdx.x * blockDim.x;
//    // id for [C/h]
//    int laneId = threadIdx.x % WARP_SIZE;
//    // id for [h*N, T]
//    //int gwid = wid + blockIdx.x * blockDim.x / WARP_SIZE;
//    //int gwid = gid / WARP_SIZE;
//
//    for (int seq_id = 0; seq_id < seq_len; seq_id++) {
//        // id for [h*N, T, C/h]
//        int gid = laneId + seq_id * row_stride + blockIdx.x * row_stride * seq_len +
//            threadIdx.y * row_stride * seq_len * batch;
//        // id for [h*N, T]
//        int gwid = seq_id + blockIdx.x * seq_len + threadIdx.y * seq_len * batch;
//        //int gwid = gid / WARP_SIZE;
//
//        float outgrad = out_grad[gid];
//        float gm = gamma[laneId];
//
//        float dlxhat = outgrad * gm;
//        float x_val = X_vals[gid];
//        float xmu = x_val - means[gwid];
//
//        float xivar = rsqrtf(vars[gwid] + 1.0e-12); // 1/tf.sqrt(x_var + epsilon)
//        float tmp = dlxhat * xmu * xivar * xivar * xivar;
//
//        // [h*N, T]
//        float dlvar = -0.5f * warpReduceSum(tmp);
//
//        float dvarx = 2.0f / row_stride * xmu;
//        float dlmu =  0.0f - dlxhat * xivar; // [h*N, T, C/h] 
//        float dlmu_sum = warpReduceSum(dlmu); // [h*N, T]
//
//        float dout = dlxhat * xivar + dlvar * dvarx + dlmu_sum / row_stride;
//
//        // [N, T, h, C/h]
//        int out_id =  laneId + blockIdx.y * row_stride + seq_id * head_num * row_stride +
//            blockIdx.x * seq_len * head_num * row_stride;
//        if (x_val < 0.0f)
//            dout *= alpha;
//        inp_grad[out_id] = dout;
//    }
//}

__global__ void LayerNormBackward2_32_8(const float* out_grad,
                                   const float* X_vals,
                                   const float* gamma,
                                   const float* vars,
                                   const float* means,
                                   float* inp_grad,
                                   int row_stride,
                                   int batch,
                                   int head_num,
                                   int seq_len,
                                   float alpha)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = (32 < row_stride ? 32 : row_stride) / WARP_SIZE;

    __shared__ float partialSum[MAX_WARP_NUM];

    out_grad += (row * row_stride);
    X_vals += (row * row_stride);
    inp_grad += (row * row_stride);

    float vals_arr;
    float gamma_reg = gamma[id];
    vals_arr = out_grad[id];
    vals_arr *= gamma_reg;

    float var_reg = vars[row];
    float mean_reg = means[row];

    float sum = 0;
    float xu;
    xu = (X_vals[id] - mean_reg);
    sum += vals_arr * xu;
    vals_arr *= rsqrtf(var_reg);

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    // 1st part: dlxhat * dxhatx
    // 2ed part: dlvar * dvarx
    vals_arr += (-sum * xu * rsqrtf(var_reg) / (var_reg));

    sum = 0;
    sum += vals_arr;

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    int seq_id = blockIdx.x % seq_len;
    int batch_id = blockIdx.x / seq_len % batch;
    int head_id = blockIdx.x / (seq_len * batch);
    inp_grad -= (row * row_stride);

    row = head_id + seq_id * head_num + batch_id * head_num * seq_len;
    inp_grad += (row * row_stride);

    float tmp = vals_arr - sum;
    // X_var doesn't need transpose as tmp is also the result before transpose
    if (X_vals[id] < 0.0f)
        tmp = alpha * tmp;
    inp_grad[id] = tmp;
}



// dishengbin
//TODO: gamma should be 1 dimension
__global__ void LayerNormBackward2(const float* out_grad,
                                   const float* X_vals,
                                   const float* gamma,
                                   const float* vars,
                                   const float* means,
                                   float* inp_grad,
                                   int row_stride,
                                   int batch,
                                   int head_num,
                                   int seq_len,
                                   float alpha)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    //int warp_num = (THREADS < row_stride ? THREADS : row_stride) / WARP_SIZE;
    int warp_num = (iteration_stride < row_stride ? iteration_stride : row_stride) / WARP_SIZE;

    __shared__ float partialSum[MAX_WARP_NUM];

    out_grad += (row * row_stride);
    X_vals += (row * row_stride);
    inp_grad += (row * row_stride);

    float vals_arr[NORM_REG];
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
    }
    // to cope with the case when row_stride cannot be divided by iteration_stride
    if ((high_index) < row_stride) {
        float gamma_reg = gamma[high_index];
        vals_arr[iterations] = out_grad[high_index];
        vals_arr[iterations] *= gamma_reg;
        iterations++;
    }

    float var_reg = vars[row];
    float mean_reg = means[row];

    float sum = 0;
    float xu[NORM_REG];
    for (int i = 0; i < iterations; i++) {
        xu[i] = (X_vals[i * iteration_stride + id] - mean_reg);
        sum += vals_arr[i] * xu[i];
        vals_arr[i] *= rsqrtf(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    // 1st part: dlxhat * dxhatx
    // 2ed part: dlvar * dvarx
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] += (-sum * xu[i] * rsqrtf(var_reg) / (var_reg));
    }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    int seq_id = blockIdx.x % seq_len;
    int batch_id = blockIdx.x / seq_len % batch;
    int head_id = blockIdx.x / (seq_len * batch);
    inp_grad -= (row * row_stride);

    row = head_id + seq_id * head_num + batch_id * head_num * seq_len;
    inp_grad += (row * row_stride);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        float tmp = vals_arr[i] - sum;

        // X_var doesn't need transpose as tmp is also the result before transpose
        if (X_vals[i * iteration_stride + id] < 0.0f)
            tmp = alpha * tmp;
        inp_grad[i * iteration_stride + id] = tmp;
    }
    if ((high_index) < row_stride) {
        float tmp = vals_arr[iterations] - sum;
        if (X_vals[high_index] < 0.0f)
            tmp = alpha * tmp;
        inp_grad[high_index] = tmp;
    }
}

__global__ void LayerNormBackward2(const half* out_grad,
                                   const half* X_vals,
                                   const half* gamma,
                                   const half* vars,
                                   const half* means,
                                   half* inp_grad,
                                   int row_stride,
                                   int batch,
                                   int head_num,
                                   int seq_len,
                                   float alpha)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = (iteration_stride < row_stride ? iteration_stride : row_stride) / WARP_SIZE;

    __shared__ float partialSum[MAX_WARP_NUM];

    half2 vals_arr[NORM_REG];
    float2 vals_arr_f[NORM_REG];

    half2* inp_grad_h = reinterpret_cast<half2*>(inp_grad);
    const half2* out_grad_h = reinterpret_cast<const half2*>(out_grad);
    const half2* vals_hat_h = reinterpret_cast<const half2*>(X_vals);

    inp_grad_h += (row * row_stride);
    out_grad_h += (row * row_stride);
    vals_hat_h += (row * row_stride);

    const half2* gamma_h = reinterpret_cast<const half2*>(gamma);
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;  // out_grad * gamma
    }
    if ((high_index) < row_stride) {
        half2 gamma_reg = gamma_h[high_index];
        vals_arr[iterations] = out_grad_h[high_index];
        vals_arr[iterations] *= gamma_reg;  // out_grad * gamma
        iterations++;
    }
    half mean_h = means[row];
    half var_h = vars[row];
    half2 var_reg = __halves2half2(var_h, var_h);
    half2 mean_reg = __halves2half2(mean_h, mean_h);
    half2 xu[NORM_REG];

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        xu[i] = (vals_hat_h[i * iteration_stride + id] - mean_reg);
        half2 result_h = (xu[i] * vals_arr[i]);
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        half2 xu_grad = ((-sum_h * xu[i] * h2rsqrt(var_reg)) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 xu_grad_f = __half22float2(xu_grad);
        vals_arr_f[i].x += xu_grad_f.x;
        vals_arr_f[i].y += xu_grad_f.y;
    }

    sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    int seq_id = blockIdx.x % seq_len;
    int batch_id = blockIdx.x / seq_len % batch;
    int head_id = blockIdx.x / (seq_len * batch);
    inp_grad_h -= (row * row_stride);
    vals_hat_h -= (row * row_stride);

    row = head_id + seq_id * head_num + batch_id * head_num * seq_len;
    inp_grad_h += (row * row_stride);
    vals_hat_h += (row * row_stride);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        half2 input = vals_hat_h[i * iteration_stride + id];
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        if (__half2float(input.x) < 0.0f)
            vals_arr_f[i].x *= alpha;
        if (__half2float(input.y) < 0.0f)
            vals_arr_f[i].y *= alpha;

        half2 temp = __float22half2_rn(vals_arr_f[i]);
        inp_grad_h[i * iteration_stride + id] = temp;
    }
    if ((high_index) < row_stride) {
        vals_arr_f[iterations].x -= sum;
        vals_arr_f[iterations].y -= sum;
        half2 temp = __float22half2_rn(vals_arr_f[iterations]);
        inp_grad_h[high_index] = temp;
    }
}

// dishengbin adopted
template <>
void launch_layerNorm_backward<float>(const float* out_grad,
                                      const float* X_data,
                                      const float* vars,
                                      const float* means,
                                      const float* gamma,
                                      float* gamma_inter,
                                      float* betta_inter,
                                      float* gamma_grad,
                                      float* betta_grad,
                                      float* inp_grad,
                                      int batch, // actually H*B*S
                                      const int hidden_dim,
                                      const int B, // batch size
                                      const int H, // head num
                                      const int S, // sequence len
                                      const float alpha, // alpha in leaky relu
                                      hipStream_t stream)
{
    int threads = 32;

    if (hidden_dim % 32 == 0) {

        if (batch > 1024) {
            dim3 grid_dim0(hidden_dim / TILE_DIM, 1024);
            dim3 block_dim0(TILE_DIM);

            LayerNormBackward1_0<float><<<grid_dim0, block_dim0, 0, stream>>>(
                    out_grad, X_data, vars, means, gamma_inter, betta_inter, batch, hidden_dim);

            dim3 grid_dim1(hidden_dim / TILE_DIM);
            dim3 block_dim1(TILE_DIM, TILE_DIM);
            LayerNormBackward1_1<float><<<grid_dim1, block_dim1, 0, stream>>>(
                    gamma_inter, betta_inter, gamma_grad, betta_grad, 1024, hidden_dim);

        }
        else {
            dim3 grid_dim1(hidden_dim / TILE_DIM);
            dim3 block_dim1(TILE_DIM, TILE_DIM);
            LayerNormBackward1<float><<<grid_dim1, block_dim1, 0, stream>>>(
                    out_grad, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);
        }


//        if (hidden_dim == 32 && batch % 8 == 0) {
//            dim3 grid_dim2(batch/8);
//            dim3 block_dim2(256); // 32 * 8
//            LayerNormBackward2_32<<<grid_dim2, block_dim2, 0, stream>>>(
//                    out_grad, X_data, gamma, vars, means, inp_grad, hidden_dim,
//                    B, H, S, alpha);
//            hipDeviceSynchronize();
//            hipError_t result = hipGetLastError();
//            if (result != hipSuccess)
//                printf("the erorr is %d in file %s at line %s\n", (int)result, __FILE__, __LINE__);
//
//        } else if (hidden_dim == 32 && batch % 4 == 0){
//            dim3 grid_dim2(batch/4);
//            dim3 block_dim2(128);
//            LayerNormBackward2_32<<<grid_dim2, block_dim2, 0, stream>>>(
//                    out_grad, X_data, gamma, vars, means, inp_grad, hidden_dim,
//                    B, H, S, alpha);
//        }
//        else
        {
            dim3 grid_dim2(batch/1);

            if (hidden_dim > 64 && hidden_dim <=128)
                threads <<= 1;
            if (hidden_dim > 128 && hidden_dim <= 256)
                threads <<= 2;
            else if (hidden_dim > 256 && hidden_dim <= 512)
                threads <<= 3;
            else if (hidden_dim > 512)
                throw std::runtime_error("Unsupport hidden_dim.");

            dim3 block_dim2(threads*1);
            LayerNormBackward2_32_8<<<grid_dim2, block_dim2, 0, stream>>>(
                    out_grad, X_data, gamma, vars, means, inp_grad, hidden_dim,
                    B, H, S, alpha);

            //dim3 grid_dim2(batch);

            //if (hidden_dim > 64 && hidden_dim <=128)
            //    threads <<= 1;
            //if (hidden_dim > 128 && hidden_dim <= 256)
            //    threads <<= 2;
            //else if (hidden_dim > 256 && hidden_dim <= 512)
            //    threads <<= 3;
            //else if (hidden_dim > 512)
            //    throw std::runtime_error("Unsupport hidden_dim.");

            //dim3 block_dim2(threads);

            //LayerNormBackward2<<<grid_dim2, block_dim2, 0, stream>>>(
            //        out_grad, X_data, gamma, vars, means, inp_grad, hidden_dim,
            //        B, H, S, alpha);

        }

    } else {
        //TODO: need to test the correctness
        dim3 grid_dim( (hidden_dim + TILE_DIM - 1) / TILE_DIM);
        dim3 block_dim(TILE_DIM, TILE_DIM);

        LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream>>>(
                out_grad, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

        dim3 grid_dim2(batch);

        if (hidden_dim > 64 && hidden_dim <=128)
            threads <<= 1;
        if (hidden_dim > 128 && hidden_dim <= 256)
            threads <<= 2;
        else if (hidden_dim > 256 && hidden_dim <= 512)
            threads <<= 3;
        else if (hidden_dim > 512)
            throw std::runtime_error("Unsupport hidden_dim.");

        dim3 block_dim2(threads);
        LayerNormBackward2<<<grid_dim2, block_dim2, 0, stream>>>(
                out_grad, X_data, gamma, vars, means, inp_grad, hidden_dim,
                B, H, S, alpha);
    }
}

// dishengbin
// TODO: for hidden_dim % 32 != 0
template <>
void launch_layerNorm_backward<half>(const half* out_grad,
                                       const half* X_data,
                                       const half* vars,
                                       const half* means,
                                       const half* gamma,
                                       half* gamma_inter,
                                       half* betta_inter,
                                       half* gamma_grad,
                                       half* betta_grad,
                                       half* inp_grad,
                                       int batch,  // actually H*B*S
                                       const int hidden_dim, // actually size_per_head
                                       const int B,      // batch size
                                       const int H,      // head num
                                       const int S,      // sequence len
                                       const float alpha, // alpha in leaky relu
                                       hipStream_t stream)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<half><<<grid_dim, block_dim, 0, stream>>>(
        out_grad, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads / 2);
    LayerNormBackward2<<<grid_dim2, block_dim2, 0, stream>>>(
        out_grad, X_data, gamma, vars, means, inp_grad, hidden_dim / 2,
        B, H, S, alpha);
}

template <typename T>
__global__ void LayerNormBackward1_fused_add(const T* __restrict__ out_grad1,
                                             const T* __restrict__ out_grad2,
                                             const T* __restrict__ vals_hat,
                                             const T* __restrict__ gamma,
                                             const T* __restrict__ betta,
                                             T* __restrict__ gamma_grad,
                                             T* __restrict__ betta_grad,
                                             int rows,
                                             int width,
                                             bool invertible)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    float betta_reg = (invertible ? (float)betta[idx] : 0.0f);
    float gamma_reg = (float)gamma[idx];

    // Loop across matrix height
    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad1[offset] + (float)out_grad2[offset];
        float val = (invertible ? ((float)vals_hat[offset] - betta_reg) / gamma_reg
                                : (float)vals_hat[offset]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        int pos = blockIdx.x * TILE_DIM + threadIdx.y;
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}

template <typename T>
__global__ void LayerNormBackward1_fused_add(const T* __restrict__ out_grad1,
                                             const T* __restrict__ out_grad2,
                                             const T* __restrict__ X_data,
                                             const T* __restrict__ vars,
                                             const T* __restrict__ means,
                                             T* __restrict__ gamma_grad,
                                             T* __restrict__ betta_grad,
                                             int rows,
                                             int width)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    int pos = blockIdx.x * TILE_DIM + threadIdx.y;
    // Loop across matrix height

    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad1[offset] + (float)out_grad2[offset];
        float val = (float)X_data[offset];
        val = (val - (float)means[r]) * rsqrtf((float)vars[r]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}

__global__ void LayerNormBackward2_fused_add(const float* out_grad1,
                                             const float* out_grad2,
                                             const float* vals_hat,
                                             const float* gamma,
                                             const float* betta,
                                             const float* vars,
                                             float* inp_grad,
                                             bool invertible,
                                             int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = (THREADS < row_stride ? THREADS : row_stride) / WARP_SIZE;
    __shared__ float partialSum[MAX_WARP_NUM];

    out_grad1 += (row * row_stride);
    out_grad2 += (row * row_stride);
    vals_hat += (row * row_stride);
    inp_grad += (row * row_stride);

    float vals_arr[NORM_REG];
    float vals_hat_arr[NORM_REG];
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad1[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] =
            (invertible ? (vals_hat[i * iteration_stride + id] - betta[i * iteration_stride + id]) /
                              gamma_reg
                        : vals_hat[i * iteration_stride + id]);
    }
    if ((high_index) < row_stride) {
        float gamma_reg = gamma[high_index];
        vals_arr[iterations] = out_grad1[high_index];
        vals_arr[iterations] *= gamma_reg;
        vals_hat_arr[iterations] =
            (invertible ? (vals_hat[high_index] - betta[high_index]) / gamma_reg
                        : vals_hat[high_index]);
        iterations++;
    }

    float var_reg = vars[row];

    float sum = 0;
    for (int i = 0; i < iterations; i++) {
        sum += vals_hat_arr[i] * vals_arr[i] * sqrtf(var_reg);
        vals_arr[i] *= rsqrtf(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) { vals_arr[i] += ((-sum * vals_hat_arr[i]) / var_reg); }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++)
        inp_grad[i * iteration_stride + id] =
            (vals_arr[i] - sum) + out_grad2[i * iteration_stride + id];
    if ((high_index) < row_stride)
        inp_grad[high_index] = (vals_arr[iterations] - sum) + out_grad2[high_index];
}

__global__ void LayerNormBackward2_fused_add(const half* out_grad1,
                                             const half* out_grad2,
                                             const half* vals_hat,
                                             const half* gamma,
                                             const half* betta,
                                             const half* vars,
                                             half* inp_grad,
                                             bool invertible,
                                             int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = (iteration_stride < row_stride ? iteration_stride : row_stride) / WARP_SIZE;
    __shared__ float partialSum[MAX_WARP_NUM];

    half2 vals_arr[NORM_REG];
    float2 vals_arr_f[NORM_REG];
    half2 vals_hat_arr[NORM_REG];

    // float2 result[iterations];

    half2* inp_grad_h = reinterpret_cast<half2*>(inp_grad);
    const half2* out_grad_h1 = reinterpret_cast<const half2*>(out_grad1);
    const half2* out_grad_h2 = reinterpret_cast<const half2*>(out_grad2);
    const half2* vals_hat_h = reinterpret_cast<const half2*>(vals_hat);

    inp_grad_h += (row * row_stride);
    out_grad_h1 += (row * row_stride);
    out_grad_h2 += (row * row_stride);
    vals_hat_h += (row * row_stride);

    const half2* gamma_h = reinterpret_cast<const half2*>(gamma);
    const half2* betta_h = (invertible ? reinterpret_cast<const half2*>(betta) : nullptr);
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h1[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;  // out_grad * gamma
        vals_hat_arr[i] =
            (invertible
                 ? (vals_hat_h[i * iteration_stride + id] - betta_h[i * iteration_stride + id]) /
                       gamma_reg
                 : vals_hat_h[i * iteration_stride + id]);
    }
    if ((high_index) < row_stride) {
        half2 gamma_reg = gamma_h[high_index];
        vals_arr[iterations] = out_grad_h1[high_index];
        vals_arr[iterations] *= gamma_reg;  // out_grad * gamma
        vals_hat_arr[iterations] =
            (invertible ? (vals_hat_h[high_index] - betta_h[high_index]) / gamma_reg
                        : vals_hat_h[high_index]);
        iterations++;
    }
    half var_h = vars[row];
    half2 var_reg = __halves2half2(var_h, var_h);

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        half2 result_h = (vals_hat_arr[i] * vals_arr[i] * h2sqrt(var_reg));
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        half2 temp = ((-sum_h * vals_hat_arr[i]) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 temp_f = __half22float2(temp);
        vals_arr_f[i].x += temp_f.x;
        vals_arr_f[i].y += temp_f.y;
    }
    sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        half2 temp = __float22half2_rn(vals_arr_f[i]);

        inp_grad_h[i * iteration_stride + id] = temp + out_grad_h2[i * iteration_stride + id];
    }
    if ((high_index) < row_stride) {
        vals_arr_f[iterations].x -= sum;
        vals_arr_f[iterations].y -= sum;
        half2 temp = __float22half2_rn(vals_arr_f[iterations]);

        inp_grad_h[high_index] = temp + out_grad_h2[high_index];
    }
}

template <>
void launch_layerNorm_backward_fused_add<float>(const float* out_grad1,
                                                const float* out_grad2,
                                                const float* vals_hat,
                                                const float* vars,
                                                const float* gamma,
                                                float* gamma_grad,
                                                float* betta_grad,
                                                float* inp_grad,
                                                int batch,
                                                int hidden_dim,
                                                hipStream_t stream,
                                                bool invertible,
                                                const float* betta)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);
    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream>>>(
        out_grad1, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads);
    LayerNormBackward2_fused_add<<<grid_dim2, block_dim2, 0, stream>>>(
        out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible, hidden_dim);
}

template <>
void launch_layerNorm_backward_fused_add<half>(const half* out_grad1,
                                                 const half* out_grad2,
                                                 const half* vals_hat,
                                                 const half* vars,
                                                 const half* gamma,
                                                 half* gamma_grad,
                                                 half* betta_grad,
                                                 half* inp_grad,
                                                 int batch,
                                                 int hidden_dim,
                                                 hipStream_t stream,
                                                 bool invertible,
                                                 const half* betta)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<half><<<grid_dim, block_dim, 0, stream>>>(
        out_grad1, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads / 2);
    LayerNormBackward2_fused_add<<<grid_dim2, block_dim2, 0, stream>>>(
        out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible, hidden_dim / 2);
}

/* Backward Normalize (Input-Gradient)
 * Using the means and variances from the input
 * This type of backward is not invertible!
 * We do the backward using the input (X)
 */

__global__ void LayerNormBackward2_fused_add(const float* out_grad1,
                                             const float* out_grad2,
                                             const float* X_vals,
                                             const float* gamma,
                                             const float* vars,
                                             const float* means,
                                             float* inp_grad,
                                             int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = (THREADS < row_stride ? THREADS : row_stride) / WARP_SIZE;
    __shared__ float partialSum[MAX_WARP_NUM];

    float vals_arr[NORM_REG];
    float vals_hat_arr[NORM_REG];

    out_grad1 += (row * row_stride);
    out_grad2 += (row * row_stride);
    X_vals += (row * row_stride);
    inp_grad += (row * row_stride);
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad1[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] = X_vals[i * iteration_stride + id];
    }
    if ((high_index) < row_stride) {
        float gamma_reg = gamma[high_index];
        vals_arr[iterations] = out_grad1[high_index];
        vals_arr[iterations] *= gamma_reg;
        vals_hat_arr[iterations] = X_vals[high_index];
        iterations++;
    }

    float var_reg = vars[row];
    float mean_reg = means[row];

    float sum = 0;
    float xu[NORM_REG];
    for (int i = 0; i < iterations; i++) {
        xu[i] = (vals_hat_arr[i] - mean_reg);
        sum += vals_arr[i] * xu[i];
        vals_arr[i] *= rsqrtf(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) {
        vals_arr[i] += (-sum * xu[i] * rsqrtf(var_reg) / (var_reg));
    }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++)
        inp_grad[i * iteration_stride + id] =
            (vals_arr[i] - sum) + out_grad2[i * iteration_stride + id];
    if ((high_index) < row_stride)
        inp_grad[high_index] = (vals_arr[iterations] - sum) + out_grad2[high_index];
}

__global__ void LayerNormBackward2_fused_add(const half* out_grad1,
                                             const half* out_grad2,
                                             const half* X_vals,
                                             const half* gamma,
                                             const half* vars,
                                             const half* means,
                                             half* inp_grad,
                                             int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = (iteration_stride < row_stride ? iteration_stride : row_stride) / WARP_SIZE;

    __shared__ float partialSum[MAX_WARP_NUM];

    half2 vals_arr[NORM_REG];
    float2 vals_arr_f[NORM_REG];
    half2 vals_hat_arr[NORM_REG];

    half2* inp_grad_h = reinterpret_cast<half2*>(inp_grad);
    const half2* out_grad_h1 = reinterpret_cast<const half2*>(out_grad1);
    const half2* out_grad_h2 = reinterpret_cast<const half2*>(out_grad2);
    const half2* vals_hat_h = reinterpret_cast<const half2*>(X_vals);

    out_grad_h1 += (row * row_stride);
    out_grad_h2 += (row * row_stride);
    inp_grad_h += (row * row_stride);
    vals_hat_h += (row * row_stride);

    const half2* gamma_h = reinterpret_cast<const half2*>(gamma);
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h1[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;  // out_grad * gamma
        vals_hat_arr[i] = vals_hat_h[i * iteration_stride + id];
    }
    if ((high_index) < row_stride) {
        half2 gamma_reg = gamma_h[high_index];
        vals_arr[iterations] = out_grad_h1[high_index];
        vals_arr[iterations] *= gamma_reg;  // out_grad * gamma
        vals_hat_arr[iterations] = vals_hat_h[high_index];
        iterations++;
    }

    half mean_h = means[row];
    half var_h = vars[row];
    half2 var_reg = __halves2half2(var_h, var_h);
    half2 mean_reg = __halves2half2(mean_h, mean_h);
    half2 xu[NORM_REG];

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        xu[i] = (vals_hat_arr[i] - mean_reg);
        half2 result_h = (xu[i] * vals_arr[i]);
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        half2 xu_grad = ((-sum_h * xu[i] * h2rsqrt(var_reg)) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 xu_grad_f = __half22float2(xu_grad);
        vals_arr_f[i].x += xu_grad_f.x;
        vals_arr_f[i].y += xu_grad_f.y;
    }

    sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        half2 temp = __float22half2_rn(vals_arr_f[i]);
        inp_grad_h[i * iteration_stride + id] = temp + out_grad_h2[i * iteration_stride + id];
    }
    if ((high_index) < row_stride) {
        vals_arr_f[iterations].x -= sum;
        vals_arr_f[iterations].y -= sum;
        half2 temp = __float22half2_rn(vals_arr_f[iterations]);
        inp_grad_h[high_index] = temp + out_grad_h2[high_index];
    }
}

template <>
void launch_layerNorm_backward_fused_add<float>(const float* out_grad1,
                                                const float* out_grad2,
                                                const float* X_data,
                                                const float* vars,
                                                const float* means,
                                                const float* gamma,
                                                float* gamma_grad,
                                                float* betta_grad,
                                                float* inp_grad,
                                                int batch,
                                                int hidden_dim,
                                                hipStream_t stream)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream>>>(
            out_grad1, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads);
    LayerNormBackward2_fused_add<<<grid_dim2, block_dim2, 0, stream>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad, hidden_dim);
}

template <>
void launch_layerNorm_backward_fused_add<half>(const half* out_grad1,
                                                 const half* out_grad2,
                                                 const half* X_data,
                                                 const half* vars,
                                                 const half* means,
                                                 const half* gamma,
                                                 half* gamma_grad,
                                                 half* betta_grad,
                                                 half* inp_grad,
                                                 int batch,
                                                 int hidden_dim,
                                                 hipStream_t stream)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<half><<<grid_dim, block_dim, 0, stream>>>(
        out_grad1, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads / 2);
    LayerNormBackward2_fused_add<<<grid_dim2, block_dim2, 0, stream>>>(
        out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad, hidden_dim / 2);
}
